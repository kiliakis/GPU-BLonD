
#include <hip/hip_runtime.h>
extern "C"
__global__ void gpu_trapz_custom(
        double *y,
        double x,
        int sz,
        double *res)
{   
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double my_sum = 0;
    for (int i = tid; i<sz-1; i+=gridDim.x*blockDim.x)
        my_sum += (y[i]+y[i+1])*x/2.0;

    atomicAdd(&(res[0]),my_sum);
}
