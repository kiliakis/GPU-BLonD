#include "hip/hip_runtime.h"

            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void gpu_copy_i2d(double *x, int *y ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = (double) y[i]*1.0;
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = (double) y[i]*1.0;
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void gpu_copy_d2d(double *x,double *y ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = y[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = y[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void gpu_complex_copy(pycuda::complex<double> *x, pycuda::complex<double> *y ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = y[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = y[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void gpu_diff(int *a, double *b, double c ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    b[i] = (a[i+1]-a[i])/c;
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    b[i] = (a[i+1]-a[i])/c;
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void set_zero_double(double *x ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = 0;
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = 0;
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void set_zero_int(int *x ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = 0;
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = 0;
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void set_zero_complex(pycuda::complex<double> *x ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = 0;
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = 0;
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void increase_by_value(double *x, double a ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] += a;
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] += a;
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void add_array(double *x, double *y ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] += y[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] += y[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void complex_mul(pycuda::complex<double> *x, pycuda::complex<double> *y, pycuda::complex<double> *z ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    z[i] = x[i] * y[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    z[i] = x[i] * y[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void gpu_mul(double *x, double *y, double a ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = a*y[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = a*y[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void gpu_copy_one(double *x, double *y, int ind ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = y[ind];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = y[ind];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void first_kernel_x(double *omega_rf, double *harmonic,  double domega_rf, int size, int counter ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    omega_rf[i*size +counter] += domega_rf * harmonic[i*size + counter] / harmonic[counter];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    omega_rf[i*size +counter] += domega_rf * harmonic[i*size + counter] / harmonic[counter];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void second_kernel_x(double *dphi_rf, double *harmonic, double *omega_rf, double *omega_rf_d, int size, int counter, double pi ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    dphi_rf[i] +=  2.0*pi*harmonic[size*i+counter]*(omega_rf[size*i+counter]-omega_rf_d[size*i+counter])/omega_rf_d[size*i+counter];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    dphi_rf[i] +=  2.0*pi*harmonic[size*i+counter]*(omega_rf[size*i+counter]-omega_rf_d[size*i+counter])/omega_rf_d[size*i+counter];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void third_kernel_x(double *x, double *y, int size_0, int counter ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i*size_0 + counter] += y[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i*size_0 + counter] += y[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void indexing_double(double *out, double *in, int *ind ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    out[i] = in[ind[i]];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    out[i] = in[ind[i]];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void indexing_int(double *out, int *in, int *ind ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    out[i] = in[ind[i]];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    out[i] = in[ind[i]];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void sincos_mul_add(double *ar, double a, double b, double *s, double *c ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    sincos(a*ar[i]+b, &s[i], &c[i]);
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    sincos(a*ar[i]+b, &s[i], &c[i]);
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void sincos_mul_add_2(double *ar, double a, double b, double *s, double *c ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    s[i] = cos(a*ar[i]+b -3.141592653589793238462643383279502884197169399375105820974944592307816406286/2); c[i] = cos(a*ar[i]+b);
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    s[i] = cos(a*ar[i]+b -3.141592653589793238462643383279502884197169399375105820974944592307816406286/2); c[i] = cos(a*ar[i]+b);
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void mul_d(double *a1, double *a2 ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    a1[i] *= a2[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    a1[i] *= a2[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void add_kernel(double *a, double *b, double *c ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    a[i]=b[i]+c[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    a[i]=b[i]+c[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void first_kernel_tracker(double *phi_rf, double x, double *phi_noise, int len, int turn ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    phi_rf[len*i + turn] += x * phi_noise[len*i + turn];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    phi_rf[len*i + turn] += x * phi_noise[len*i + turn];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void second_kernel_tracker(double *phi_rf, double *omega_rf, double *phi_mod0, double *phi_mod1, int size, int turn ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    phi_rf[i*size+turn] += phi_mod0[i*size+turn]; omega_rf[i*size+turn] += phi_mod1[i*size+turn];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    phi_rf[i*size+turn] += phi_mod0[i*size+turn]; omega_rf[i*size+turn] += phi_mod1[i*size+turn];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void copy_column(double *x, double *y, int size, int column ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = y[i*size + column];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = y[i*size + column];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void rf_voltage_calculation_kernel(double *x, double *y, int size, int column ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    x[i] = y[i*size + column];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    x[i] = y[i*size + column];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void cavityFB_case(double *rf_voltage, double *voltage, double *omega_rf, double *phi_rf,double *bin_centers, double V_corr, double phi_corr,int size, int column ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    rf_voltage[i] = voltage[0] * V_corr * sin(omega_rf[0] * bin_centers[i]+phi_rf[0]+phi_corr);
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    rf_voltage[i] = voltage[0] * V_corr * sin(omega_rf[0] * bin_centers[i]+phi_rf[0]+phi_corr);
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void bm_phase_exp_times_scalar(double *a, double *b, double c, int *d ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    a[i] = exp(c*b[i])*d[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    a[i] = exp(c*b[i])*d[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void bm_phase_mul_add(double *a, double b, double *c, double d ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    a[i] = b*c[i] + d;
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    a[i] = b*c[i] + d;
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void bm_sin_cos(double *a, double *b, double *c ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    sincos(a[i],&b[i], &c[i]);
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    sincos(a[i],&b[i], &c[i]);
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void d_multiply(double *a, double *b ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    a[i] *= b[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    a[i] *= b[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void d_multscalar(double *a, double *b, double c ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    a[i] = c*b[i];
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    a[i] = c*b[i];
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void scale_kernel_int(int a, int *b ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    b[i] /= a ;
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    b[i] /= a ;
                }
                }
                ;
            }
            
            #include <pycuda-complex.hpp>
            
            extern "C"
            __global__ void scale_kernel_double(double a, double *b ,long start, long stop, long step)
            {
                unsigned tid = threadIdx.x;
                unsigned total_threads = gridDim.x*blockDim.x;
                unsigned cta_start = blockDim.x*blockIdx.x;
                long i;
                ;
                if (step < 0)
                {
                for (i = start + (cta_start + tid)*step;
                    i > stop; i += total_threads*step)
                {
                    b[i] /= a ;
                }
                }
                else
                {
                for (i = start + (cta_start + tid)*step;
                    i < stop; i += total_threads*step)
                {
                    b[i] /= a ;
                }
                }
                ;
            }
            