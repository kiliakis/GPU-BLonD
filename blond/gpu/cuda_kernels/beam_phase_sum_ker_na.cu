
#include <hip/hip_runtime.h>
extern "C"
__global__ void beam_phase_sum(
    const double *ar1,
    const double *ar2,
    double *scoeff,
    double *coeff,
    int n_bins)
{   
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid==0){
        scoeff[0]=0;
        coeff[0] =0;
    }
    double my_sum_1 = 0;
    double my_sum_2 = 0;
    if (tid==0){
        my_sum_1 += ar1[0]/2+ar1[n_bins-1]/2;
        my_sum_2 += ar2[0]/2+ar2[n_bins-1]/2;
    }
    for (int i = tid+1; i<n_bins-1; i+=gridDim.x*blockDim.x){
        my_sum_1 += ar1[i];
        my_sum_2 += ar2[i];
    }
    atomicAdd(&(scoeff[0]),my_sum_1);
    atomicAdd(&(coeff[0]),my_sum_2);
    __syncthreads();
    if (tid==0)
        scoeff[0]=scoeff[0]/coeff[0];
    
}